#include "hip/hip_runtime.h"
#include "barnes_hut_force.h"
#include "particle_utils.h"
#include <chrono>
#include <iostream>

/**
 * Barnes-Hut force calculation implementation
 * This replaces O(N²) with O(N log N) using octree approximation
 */

BarnesHutForce::BarnesHutForce(float theta, float softening)
    : theta(theta), softening(softening), last_force_calc_time(0.0) {

  // Create CUDA events for timing
  checkCudaError(hipEventCreate(&start_event),
                 "create start event for Barnes-Hut");
  checkCudaError(hipEventCreate(&stop_event),
                 "create stop event for Barnes-Hut");

  std::cout << "Barnes-Hut force calculator initialized:" << std::endl;
  std::cout << "  Theta (approximation parameter): " << theta << std::endl;
  std::cout << "  Softening factor: " << softening << std::endl;
}

void BarnesHutForce::calculateForces(ParticleSystem &particles,
                                     const Octree &octree, float G_constant) {
  int N = particles.getNumParticles();
  if (N == 0 || octree.getNumNodes() == 0) {
    std::cout << "Warning: No particles or nodes for force calculation"
              << std::endl;
    return;
  }

  // Record start time
  checkCudaError(hipEventRecord(start_event), "record Barnes-Hut start event");

  // Extract particle data
  Particle *device_particles = particles.getDeviceParticles();

  // Allocate temporary arrays for positions, masses, and accelerations
  float3 *d_positions;
  float *d_masses;
  float3 *d_accelerations;

  checkCudaError(hipMalloc(&d_positions, N * sizeof(float3)),
                 "positions for Barnes-Hut");
  checkCudaError(hipMalloc(&d_masses, N * sizeof(float)),
                 "masses for Barnes-Hut");
  checkCudaError(hipMalloc(&d_accelerations, N * sizeof(float3)),
                 "accelerations for Barnes-Hut");

  // Extract data from the main particle buffer into temporary flat arrays
  launch_extract_positions(device_particles, d_positions, N);
  launch_extract_masses(device_particles, d_masses, N);

  // Reset accelerations to zero before accumulation
  launch_reset_accelerations_bh(d_accelerations, N);

  // Calculate forces using Barnes-Hut algorithm
  launch_barnes_hut_force_calculation(
      d_positions, d_masses, d_accelerations, N, octree.getDeviceNodes(),
      octree.getNumNodes(), octree.getDeviceParticleIndices(),
      theta * theta,         // theta squared for efficiency
      softening * softening, // softening squared
      G_constant);

  // The new kernel computes accelerations in the original particle order.
  // Therefore, we use the simple update kernel, not the reordered one.
  launch_update_accelerations(device_particles, d_accelerations, N);

  // Record end time
  checkCudaError(hipEventRecord(stop_event), "record Barnes-Hut stop event");
  checkCudaError(hipEventSynchronize(stop_event),
                 "synchronize Barnes-Hut stop event");

  // Calculate elapsed time
  float elapsed_time_ms;
  checkCudaError(
      hipEventElapsedTime(&elapsed_time_ms, start_event, stop_event),
      "get Barnes-Hut elapsed time");
  last_force_calc_time = elapsed_time_ms / 1000.0; // Convert to seconds

  // Clean up
  hipFree(d_positions);
  hipFree(d_masses);
  hipFree(d_accelerations);
}

void BarnesHutForce::printPerformanceStats() const {
  std::cout << "\n=== Barnes-Hut Performance Statistics ===" << std::endl;
  std::cout << "Last force calculation time: " << last_force_calc_time * 1000.0
            << " ms" << std::endl;
  std::cout << "Theta parameter: " << theta << std::endl;
  std::cout << "Softening factor: " << softening << std::endl;
  std::cout << "======================================\n" << std::endl;
}

/**
 * CUDA kernels for Barnes-Hut force calculation
 */

// Kernel to reset accelerations
__global__ void reset_accelerations_kernel(float3 *accelerations, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    accelerations[idx] = make_float3(0.0f, 0.0f, 0.0f);
  }
}

// Main Barnes-Hut force calculation kernel
__global__ void
barnes_hut_force_kernel(const float3 *positions, const float *masses,
                        float3 *accelerations, int N, const OctreeNode *nodes,
                        int num_nodes, const int *particle_indices,
                        float theta_sq, float softening_sq, float G_constant) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= N)
    return;

  float3 my_pos = positions[i];
  float my_mass = masses[i];
  float3 acc = make_float3(0.0f, 0.0f, 0.0f);

  TraversalStack stack;
  if (num_nodes > 0) {
    stack.push(0); // Start traversal at the root (node 0)
  }

  while (!stack.isEmpty()) {
    int node_idx = stack.pop();
    if (node_idx < 0 || node_idx >= num_nodes)
      continue;

    const OctreeNode &node = nodes[node_idx];

    // Vector from particle i to node's center of mass
    float3 r_vec = node.center_of_mass - my_pos;
    float dist_sq = dot(r_vec, r_vec);

    if (node.isLeaf()) {
      // It's a leaf node. Iterate over the particles in this leaf.
      for (int k = 0; k < node.particle_count; ++k) {
        int sorted_particle_idx = node.particle_start_idx + k;
        if (sorted_particle_idx >= N)
          continue; // Boundary check

        int original_particle_idx = particle_indices[sorted_particle_idx];
        if (original_particle_idx == i)
          continue; // Skip self-interaction

        // Direct particle-particle interaction
        float3 p_pos = positions[original_particle_idx];
        float3 r_vec_p = p_pos - my_pos;
        float p_dist_sq = dot(r_vec_p, r_vec_p) + softening_sq;

        if (p_dist_sq > 1e-9f) {
          float inv_dist = rsqrtf(p_dist_sq);
          float inv_dist_cubed = inv_dist * inv_dist * inv_dist;
          float p_mass = masses[original_particle_idx];
          float force_scalar = G_constant * my_mass * p_mass * inv_dist_cubed;
          acc += r_vec_p * force_scalar;
        }
      }
    } else {
      // It's an internal node. Check the Barnes-Hut criterion.
      float3 node_size_vec = node.getSize();
      float node_size_sq =
          dot(node_size_vec, node_size_vec); // Using squared size

      // s/d < theta  -> s^2 < d^2 * theta^2
      if (node_size_sq < dist_sq * theta_sq) {
        // Node is far enough away, approximate it as a single mass.
        float dist_with_softening_sq = dist_sq + softening_sq;
        if (dist_with_softening_sq > 1e-9f) {
          float inv_dist = rsqrtf(dist_with_softening_sq);
          float inv_dist_cubed = inv_dist * inv_dist * inv_dist;
          float force_mag =
              G_constant * my_mass * node.total_mass * inv_dist_cubed;
          acc += r_vec * force_mag;
        }
      } else {
        // Node is too close, traverse its children.
        if (node.first_child_idx >= 0) {
          for (int child = 0; child < 8; ++child) {
            int child_idx = node.first_child_idx + child;
            if (child_idx < num_nodes && nodes[child_idx].particle_count > 0) {
              if (!stack.isFull()) {
                stack.push(child_idx);
              }
            }
          }
        }
      }
    }
  }

  // Final acceleration is F/m
  if (my_mass > 0.0f) {
    acc = acc * (1.0f / my_mass);
  } else {
    acc = make_float3(0.0f, 0.0f, 0.0f);
  }
  accelerations[i] = acc;
}

/**
 * Wrapper functions for kernel launches
 */
extern "C" void launch_reset_accelerations_bh(float3 *accelerations, int N) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  reset_accelerations_kernel<<<blocksPerGrid, threadsPerBlock>>>(accelerations,
                                                                 N);
  checkCudaError(hipGetLastError(), "reset accelerations kernel launch");
}

extern "C" void launch_barnes_hut_force_calculation(
    const float3 *positions, const float *masses, float3 *accelerations, int N,
    const OctreeNode *nodes, int num_nodes, const int *particle_indices,
    float theta_sq, float softening_sq, float G_constant) {

  // This kernel launch is simplified because we are not reordering inside.
  // The reordering is handled by the data extraction and final update steps.
  // Therefore, the kernel can operate on flat, unsorted arrays directly.
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  barnes_hut_force_kernel<<<blocksPerGrid, threadsPerBlock>>>(
      positions, masses, accelerations, N, nodes, num_nodes, particle_indices,
      theta_sq, softening_sq, G_constant);
}