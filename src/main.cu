#include "initial_conditions.h"
#include "nbody_simulation.h"
#include <iostream>
#include <vector>

// Forward declare the function that will run the GUI
void run_simulation_with_gui(NBodySimulation &sim);

void runSimpleDemo() {
  const int N = 8192;
  std::cout << "\n--- Running Simple Sphere Demo with Visualization (N=" << N
            << ") ---" << std::endl;

  NBodySimulation sim(N, NBodySimulation::BARNES_HUT);
  InitialConditions::generateRandomSphere(*sim.getParticleSystem(), 5.0f, 1.0f);

  sim.getParticleSystem()->copyToDevice();

  run_simulation_with_gui(sim);
}

void runGalaxyDemo() {
  std::cout << "\n=== Galaxy Simulation Demo ===" << std::endl;

  const int N = 2000;
  const int timesteps = 200;

  // Create simulation with smaller timestep for stability
  NBodySimulation sim(N, NBodySimulation::BARNES_HUT, 0.005f, 0.05f);

  // Generate galaxy disk
  InitialConditions::generateGalaxyDisk(*sim.getParticleSystem(), 8.0f, 0.5f,
                                        100.0f, 2.0f);

  // Copy to device
  sim.getParticleSystem()->copyToDevice();

  // Print initial statistics
  sim.getParticleSystem()->printStatistics();

  // Run simulation
  sim.simulate(timesteps);

  // Copy back to host and print final statistics
  sim.getParticleSystem()->copyToHost();
  sim.getParticleSystem()->printStatistics();
}

void runCollidingClustersDemo() {
  std::cout << "\n=== Colliding Clusters Demo ===" << std::endl;

  const int N = 1500;
  const int timesteps = 300;

  // Create simulation
  NBodySimulation sim(N, NBodySimulation::BARNES_HUT, 0.008f, 0.08f);

  // Generate colliding clusters
  InitialConditions::generateCollidingClusters(*sim.getParticleSystem(), 15.0f,
                                               3.0f, 1.5f);

  // Copy to device
  sim.getParticleSystem()->copyToDevice();

  // Print initial statistics
  sim.getParticleSystem()->printStatistics();

  // Run simulation
  sim.simulate(timesteps);

  // Copy back to host and print final statistics
  sim.getParticleSystem()->copyToHost();
  sim.getParticleSystem()->printStatistics();
}

void runBenchmark() {
  std::cout << "\n=== Performance Benchmark ===" << std::endl;

  // Test different particle counts
  std::vector<int> particle_counts = {512, 1024, 2048, 4096, 8192, 16384};
  const int timesteps_per_test = 10;

  // Create a simulation instance for benchmarking
  NBodySimulation benchmark_sim(particle_counts[0], NBodySimulation::BARNES_HUT,
                                0.01f, 0.1f);

  // Run benchmark
  benchmark_sim.benchmark(particle_counts, timesteps_per_test);

  std::cout << "\nBenchmark completed!" << std::endl;
  std::cout << "Note: Performance will vary based on GPU memory and compute "
               "capability."
            << std::endl;
  std::cout << "Your RTX 3060 should achieve significant speedup over CPU "
               "implementations!"
            << std::endl;
}

void printUsage(const char *program_name) {
  std::cout << "Usage: " << program_name << " [option]" << std::endl;
  std::cout << "Options:" << std::endl;
  std::cout << "  simple     - Run simple random sphere demo (1000 particles)"
            << std::endl;
  std::cout << "  galaxy     - Run galaxy disk demo (2000 particles)"
            << std::endl;
  std::cout << "  collision  - Run colliding clusters demo (1500 particles)"
            << std::endl;
  std::cout << "  benchmark  - Run performance benchmark" << std::endl;
  std::cout << "  all        - Run all demos and benchmark (default)"
            << std::endl;
}

int main(int argc, char *argv[]) {
  std::cout << "===============================================" << std::endl;
  std::cout << "    GPU N-Body Simulation - Phase 4: Visualization"
            << std::endl;
  std::cout << "===============================================" << std::endl;

  // Check CUDA device
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    std::cerr << "No CUDA devices found!" << std::endl;
    return 1;
  }

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  std::cout << "Using GPU: " << deviceProp.name << std::endl;
  std::cout << "Compute Capability: " << deviceProp.major << "."
            << deviceProp.minor << std::endl;
  std::cout << "Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024)
            << " MB" << std::endl;
  std::cout << "Multiprocessors: " << deviceProp.multiProcessorCount
            << std::endl;
  std::cout << std::endl;

  // For now, we will just run the simple demo.
  // Command-line parsing can be re-added later.
  try {
    runSimpleDemo();
  } catch (const std::exception &e) {
    std::cerr << "An error occurred: " << e.what() << std::endl;
    return 1;
  }

  std::cout << "\n===============================================" << std::endl;
  std::cout << "    Simulation Complete" << std::endl;
  std::cout << "===============================================" << std::endl;

  std::cout << "\nPress any key to continue..." << std::endl;
  std::cin.get();

  return 0;
}